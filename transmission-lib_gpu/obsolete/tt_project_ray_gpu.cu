#include "hip/hip_runtime.h"

// Utilities and System includes
#include <cutil_inline.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <driver_functions.h>
#include <tt_project_ray_gpu.h>
#include <_tt_common.h>
#include <sys/time.h>

#define MAX_EPSILON_ERROR 5.00f
#define THRESHOLD         0.30f

#define MAX(a,b) ((a > b) ? a : b)

extern "C" void setTextureFilterMode(bool bLinearFilter);
extern "C" void initCuda(void *h_volume, hipExtent volumeSize);
extern "C" void freeCudaBuffers();
extern "C" void tt_project_ray_kernel(dim3 gridSize, dim3 blockSize, float *d_output, float3 source_position, float3 volume_size, u_int imageW, u_int imageH, float t_step);
extern "C" void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix);


int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

int set_inViewMatrix(float *invViewMatrix, float_2 detector_scale, float_3 detector_transl, float_3 detector_rotat)
{
    memset((void*)invViewMatrix,0,12*sizeof(float));
    //rotate
    mat44 *rotation = (mat44 *)calloc(1,sizeof(mat44));
    create_rotation_matrix44(rotation, detector_rotat.x,detector_rotat.y,detector_rotat.z,0,0,0);
    //scale
    mat44 *scale = (mat44 *)calloc(1,sizeof(mat44));
    scale->m[0][0] =detector_scale.x;
    scale->m[1][1] =detector_scale.y;
    scale->m[2][2] =1;
    //transform
    mat44 *m = (mat44 *)calloc(1,sizeof(mat44));
    *m = reg_mat44_mul(rotation,scale);
    invViewMatrix[0]=m->m[0][0]; invViewMatrix[1]=m->m[0][1]; invViewMatrix[2] =m->m[0][2]; 
    invViewMatrix[4]=m->m[1][0]; invViewMatrix[5]=m->m[1][1]; invViewMatrix[6] =m->m[1][2]; 
    invViewMatrix[8]=m->m[2][0]; invViewMatrix[9]=m->m[2][1]; invViewMatrix[10]=m->m[2][2];
    //translate
    invViewMatrix[3] =detector_transl.x;
    invViewMatrix[7] =detector_transl.y;
    invViewMatrix[11]=detector_transl.z; 
    //cleanup
    free(rotation);
    free(scale);
    free(m);
    return 0;
}


////////////////////////////////////////////////////////////////////////////////
// tt_project_ray_array
////////////////////////////////////////////////////////////////////////////////
extern "C" int tt_project_ray_array(VolumeType h_volume[], u_int_3 volume_voxels, float out_projections[], u_int n_projections, float_2 detector_scale[], float_3 detector_transl[], float_3 detector_rotat[], u_int_2 detector_pixels, float_3 source_position[], float_3 volume_size, float t_step)
{
    dim3 blockSize(16, 16);
    bool linearFiltering = true;
    dim3 gridSize;

    float  invViewMatrix[12];
    float3 sourcePosition;
    float3 volumeSize;
    float *d_output;
    float *d_output_proj;

    hipInit(0);

    hipExtent vsize = make_hipExtent(volume_voxels.x,volume_voxels.y,volume_voxels.z);
fprintf(stderr, "\n %d, %d, %d",volume_voxels.x,volume_voxels.y,volume_voxels.z);
    initCuda(h_volume, vsize);
    setTextureFilterMode(linearFiltering);
    gridSize = dim3(iDivUp(detector_pixels.w, blockSize.x), iDivUp(detector_pixels.h, blockSize.y));

    //Allocate memory for projections on the device
    CUDA_SAFE_CALL(hipMalloc((void **)&d_output, n_projections*detector_pixels.w*detector_pixels.h*sizeof(float) ));
    CUDA_SAFE_CALL(hipMemset((void *)d_output,0, n_projections*detector_pixels.w*detector_pixels.h*sizeof(float) ));

    struct timeval start_time; gettimeofday( &start_time, 0);
    struct timeval t_time;
    float elapsed_time;

    volumeSize.x = volume_size.x;
    volumeSize.y = volume_size.y;
    volumeSize.z = volume_size.z;

    for (int proj=0;proj<n_projections;proj++)
    {
        //define invViewMatrix (position of detector) and position of source
        set_inViewMatrix(invViewMatrix, detector_scale[proj], detector_transl[proj], detector_rotat[proj]);
        sourcePosition.x = source_position[proj].x;
        sourcePosition.y = source_position[proj].y;
        sourcePosition.z = source_position[proj].z;

        //project
        copyInvViewMatrix(invViewMatrix, sizeof(float4)*3);
        d_output_proj = (float*) d_output + proj * detector_pixels.w * detector_pixels.h;
        tt_project_ray_kernel(gridSize, blockSize, d_output_proj, sourcePosition, volumeSize, detector_pixels.w, detector_pixels.h, t_step);
    }

    gettimeofday( &t_time, 0);
    elapsed_time = (float) (1000.0 * ( t_time.tv_sec - start_time.tv_sec) + (0.001 * (t_time.tv_usec - start_time.tv_usec)) );
    fprintf(stderr,"\nTime per projection %d %d %d -> %d %d: %f ms",volume_voxels.x,volume_voxels.y,volume_voxels.z,detector_pixels.w,detector_pixels.h,elapsed_time/n_projections);

    //Copy result back to host
    CUDA_SAFE_CALL(hipMemcpy(out_projections, d_output, n_projections*detector_pixels.w*detector_pixels.h*sizeof(float), hipMemcpyDeviceToHost));

    //Clean up
    CUDA_SAFE_CALL(hipFree(d_output));
    freeCudaBuffers();
    hipDeviceReset();
    return 0;
}

