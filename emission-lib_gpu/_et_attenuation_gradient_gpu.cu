#include "hip/hip_runtime.h"
/*
 *  _et_attenuation_gradient_gpu.cu
 *  
 *  NiftyRec
 *  Stefano Pedemonte, May 2012.
 *  CMIC - Centre for Medical Image Computing 
 *  UCL - University College London. 
 *  Released under BSD licence, see LICENSE.txt 
 */

#include "_et_attenuation_gradient_gpu.h"
#include "_et_attenuation_gradient_gpu_kernels.cu"

void et_attenuation_gradient_gpu(float **d_activity, float **d_sinogram, float **d_backprojection, float **d_attenuation, int cam, nifti_image *backprojection)
{
	int3 backprojection_size = make_int3(backprojection->nx,backprojection->ny,backprojection->nz);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_backprojection_size),&backprojection_size,sizeof(int3)));
	
	const unsigned int grid = (unsigned int)ceil(backprojection->nx*backprojection->ny/(float)BLOCK);
	dim3 B(BLOCK,1,1);
	dim3 G(grid,1,1);
	
	float *d_sinogram_ptr = (*d_sinogram) + cam * backprojection->nx * backprojection->ny;
	
	et_attenuation_gradient_gpu_kernel <<<G,B>>> (*d_activity, d_sinogram_ptr, *d_backprojection, *d_attenuation);
	
	CUDA_SAFE_CALL(hipDeviceSynchronize());
}



