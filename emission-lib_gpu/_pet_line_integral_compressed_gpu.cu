#include "hip/hip_runtime.h"
/*
 *  _pet_line_integrals_compressed_gpu.cu
 *  
 *  NiftyRec
 *  Stefano Pedemonte, Oct. 2012.
 *  CMIC - Centre for Medical Image Computing 
 *  UCL - University College London. 
 *  Harvard University, Martinos Center for Biomedical Imaging
 *  Jan. 2014.
 */

#include "_pet_line_integral_compressed_gpu.h"
#include "_pet_line_integral_compressed_gpu_kernels.cu"


void pet_line_integral_compressed_gpu(float *d_activity, float *d_attenuation, float *d_projection, unsigned short *d_locations, unsigned int N_locations, unsigned int N_u, unsigned int N_v, unsigned int N_samples, unsigned int direction, unsigned int block_size)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_N_locations), &N_locations,sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_N_u),         &N_u,sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_N_v),         &N_v,sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_N_samples),   &N_samples,sizeof(unsigned int)));

    const unsigned int Grid = (unsigned int)ceil((float)N_locations/(float)block_size); 
    
    // note: block_size affects the performance, a good value is 512
    
	dim3 B1(block_size,1,1);
	dim3 G1(Grid,1,1);

	pet_line_integral_compressed_gpu_kernel <<<G1,B1>>> (d_activity, d_attenuation, d_projection, d_locations, direction);

	CUDA_SAFE_CALL(hipDeviceSynchronize());
} 


