/*
 *  _et_convolveFFT2D_gpu.cu
 *  
 *  NiftyRec
 *  Stefano Pedemonte, May 2012.
 *  CMIC - Centre for Medical Image Computing 
 *  UCL - University College London. 
 *  Released under BSD licence, see LICENSE.txt 
 */

#include "_et_convolveFFT2D_gpu_kernels.cu"
#include "_et_common.h"

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
int calculateFFTsize(int dataSize){
    int hiBit;
    unsigned int lowPOT, hiPOT;

    dataSize = iAlignUp(dataSize, 16);

    for(hiBit = 31; hiBit >= 0; hiBit--)
        if(dataSize & (1U << hiBit)) break;

    lowPOT = 1U << hiBit;
    if(lowPOT == dataSize)
        return dataSize;

    hiPOT = 1U << (hiBit + 1);
    if(hiPOT <= 1024)
        return hiPOT;
    else 
        return iAlignUp(dataSize, 512);
}



////////////////////////////////////////////////////////////////////////////////
// 2D Convolution
////////////////////////////////////////////////////////////////////////////////

int et_convolveFFT2D_gpu(float **d_data, int *data_size, float **d_kernel, int *kernel_size, float **d_result)
{
    int status = 1;
    const int dataH = data_size[0];
    const int dataW = data_size[1];
    const int kernelH = kernel_size[0];
    const int kernelW = kernel_size[1];

    const int kernelX = (kernelH-1)/2;
    const int kernelY = (kernelW-1)/2;

    const int n_slices = data_size[2];
    const int data_slice_size = dataH * dataW;
    const int kernel_slice_size = kernelH * kernelW;

    float *d_PaddedData, *d_PaddedKernel, *d_Data, *d_Kernel, *d_Result;
    fComplex *d_DataSpectrum, *d_KernelSpectrum;
    hipfftHandle fftPlanFwd, fftPlanInv;

    //Derive FFT size from data and kernel dimensions
    const int fftW = calculateFFTsize(dataW + kernelW - 1);
    const int fftH = calculateFFTsize(dataH + kernelH - 1);

    //Allocate memory for zero-padded image and kernel and for their transforms
//    fprintf_verbose("Allocating memory...\n");
    CUDA_SAFE_CALL( hipMalloc((void **)&d_PaddedKernel, fftH * fftW * sizeof(float)) );
    CUDA_SAFE_CALL( hipMalloc((void **)&d_PaddedData,   fftH * fftW * sizeof(float)) );

    CUDA_SAFE_CALL( hipMalloc((void **)&d_KernelSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)) );
    CUDA_SAFE_CALL( hipMalloc((void **)&d_DataSpectrum,   fftH * (fftW / 2 + 1) * sizeof(fComplex)) );

    //Create cuFFT plan
//    fprintf_verbose("Creating FFT plan for %i x %i...\n", fftH, fftW);
    CUDA_SAFE_CALL( hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C) );
    CUDA_SAFE_CALL( hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R) );

    //Convolve slices one by one
    for (int slice=0; slice<n_slices; slice++)
        {
        //Determine slice pointer
        d_Data = (*d_data) + slice * data_slice_size; 
        d_Kernel = (*d_kernel) + slice * kernel_slice_size;
        d_Result = (*d_result) + slice * data_slice_size;

        //Zero pad
//        fprintf_verbose("Padding convolution kernel and input data...\n");
        CUDA_SAFE_CALL( hipMemset(d_PaddedKernel, 0, fftH * fftW * sizeof(float)) );
        CUDA_SAFE_CALL( hipMemset(d_PaddedData,   0, fftH * fftW * sizeof(float)) );
        padKernel(d_PaddedKernel,d_Kernel,fftH,fftW,kernelH,kernelW,kernelY,kernelX);
CUDA_SAFE_CALL( hipDeviceSynchronize() );
	if (!d_PaddedData || !d_PaddedKernel) fprintf_verbose("NULL arguments!\n");
//        fprintf_verbose( "%d %d %d %d %d %d %d %d %d %d\n", d_PaddedData,d_Data,fftH,fftW,dataH,dataW,kernelH,kernelW,kernelY,kernelX);
        padDataClampToBorder(d_PaddedData,d_Data,fftH,fftW,dataH,dataW,kernelH,kernelW,kernelY,kernelX);
CUDA_SAFE_CALL( hipDeviceSynchronize() );
        //Convolve
//        fprintf_verbose("Transforming convolution kernel...\n");
        CUDA_SAFE_CALL( hipfftExecR2C(fftPlanFwd, d_PaddedKernel, (hipfftComplex *)d_KernelSpectrum) );

//        fprintf_verbose("Running GPU FFT convolution...\n");
        CUDA_SAFE_CALL( hipDeviceSynchronize() );
        CUDA_SAFE_CALL( hipfftExecR2C(fftPlanFwd, d_PaddedData, (hipfftComplex *)d_DataSpectrum) );
        modulateAndNormalize(d_DataSpectrum, d_KernelSpectrum, fftH, fftW);
        CUDA_SAFE_CALL( hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_DataSpectrum, d_PaddedData) );
        CUDA_SAFE_CALL( hipDeviceSynchronize() );
      

        //Crop result
//        fprintf_verbose("Cropping result image...\n");
        //CUDA_SAFE_CALL( hipMemset(d_Result, 11, dataH * dataW * sizeof(float)) ); //FIXME do the real thing
        crop_image(d_Result,d_PaddedData,fftH,fftW,dataH,dataW,kernelH,kernelW);
        }

    //Destroy cuFFT plan and free memory
//    fprintf_verbose("Shutting down...\n");
    CUDA_SAFE_CALL( hipfftDestroy(fftPlanInv) );
    CUDA_SAFE_CALL( hipfftDestroy(fftPlanFwd) );
    CUDA_SAFE_CALL( hipFree(d_DataSpectrum)   );
    CUDA_SAFE_CALL( hipFree(d_KernelSpectrum) );
    CUDA_SAFE_CALL( hipFree(d_PaddedData)   );
    CUDA_SAFE_CALL( hipFree(d_PaddedKernel) );

    status = 0;
    return status;
}



