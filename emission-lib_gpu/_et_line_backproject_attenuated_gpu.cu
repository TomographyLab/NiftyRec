#include "hip/hip_runtime.h"
/*
 *  _et_line_backproject_attenuated_gpu.cu
 *  
 *  NiftyRec
 *  Stefano Pedemonte, May 2012.
 *  CMIC - Centre for Medical Image Computing 
 *  UCL - University College London. 
 *  Released under BSD licence, see LICENSE.txt 
 */

#include "_et_line_backproject_attenuated_gpu.h"
#include "_et_line_backproject_attenuated_gpu_kernels.cu"

void et_line_backproject_attenuated_gpu(float **d_sinogram, float **d_backprojection, float **d_attenuation, int cam, nifti_image *backprojection)
{
	int3 backprojection_size = make_int3(backprojection->nx,backprojection->ny,backprojection->nz);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_backprojection_size),&backprojection_size,sizeof(int3)));
	
	const unsigned int grid = (unsigned int)ceil(backprojection->nx*backprojection->ny/(float)BLOCK);
	dim3 B(BLOCK,1,1);
	dim3 G(grid,1,1);
	
	float *d_sinogram_ptr = (*d_sinogram) + cam * backprojection->nx * backprojection->ny;
	
	et_line_backproject_attenuated_gpu_kernel <<<G,B>>> (d_sinogram_ptr, *d_backprojection, *d_attenuation);
	
	CUDA_SAFE_CALL(hipDeviceSynchronize());
}



