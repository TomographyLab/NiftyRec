#include "hip/hip_runtime.h"
/*
 *  _et_joint_histogram_gpu_kernels.cu
 *  
 *  NiftyRec
 *  Stefano Pedemonte, May 2012.
 *  CMIC - Centre for Medical Image Computing 
 *  UCL - University College London. 
 *  Released under BSD licence, see LICENSE.txt 
 */

#include "_et_joint_histogram_gpu.h"

__global__ void et_joint_histogram_gpu_kernel(float *d_array_A, float *d_array_B, int *d_joint_hist, int array_size, int hist_size, float min_A, float max_A, float min_B, float max_B)
{
	__shared__ int s_binned_A[BLOCK];
	__shared__ int s_binned_B[BLOCK];
	
	const unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;

	float scale_A = (float)hist_size/(max_A-min_A);
	float scale_B = (float)hist_size/(max_B-min_B);

	if(tid < array_size){
		//load array to shared mem and bin it
		s_binned_A[threadIdx.x] = floor( (d_array_A[tid] - min_A) * scale_A );
		s_binned_B[threadIdx.x] = floor( (d_array_B[tid] - min_B) * scale_B );

		if (s_binned_A[threadIdx.x] < 0)
			s_binned_A[threadIdx.x] = 0;
		if (s_binned_B[threadIdx.x] < 0)
			s_binned_B[threadIdx.x] = 0;
		if (s_binned_A[threadIdx.x] >= hist_size)
			s_binned_A[threadIdx.x] = hist_size-1;
		if (s_binned_B[threadIdx.x] >= hist_size)
			s_binned_B[threadIdx.x] = hist_size-1;

		__syncthreads();
	}
	//only one thread writes the results
	if (threadIdx.x == 0)
		for(int i=0;i< blockDim.x; i++)
			if (blockIdx.x*blockDim.x + i < array_size)
				d_joint_hist[ s_binned_A[i] * hist_size + s_binned_B[i] ] ++;
	//d_joint_hist[ s_binned_A[threadIdx.x] * hist_size + s_binned_B[threadIdx.x] ] ++;
	return;  
}





