#include "hip/hip_runtime.h"
/*
 *  _et_line_integrals_gpu.cu
 *  
 *  NiftyRec
 *  Stefano Pedemonte, May 2012.
 *  CMIC - Centre for Medical Image Computing 
 *  UCL - University College London. 
 *  Released under BSD licence, see LICENSE.txt 
 */

#include "_et_line_integral_gpu.h"
#include "_et_line_integral_gpu_kernels.cu"

#define BLOCK 256

void et_line_integral_gpu(float *d_activity, float *d_sinogram, int cam, nifti_image *img, float background_activity)
{
	int3 imageSize = make_int3(img->dim[1],img->dim[2],img->dim[3]);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ImageSize),&imageSize,sizeof(int3)));
	
	const unsigned int Grid = (unsigned int)ceil(img->dim[1]*img->dim[2]/(float)BLOCK);
	dim3 B1(BLOCK,1,1);
	dim3 G1(Grid,1,1);
	
	float *currentCamPointer = (d_sinogram) + cam * img->dim[1] * img->dim[2] ;
	
	et_line_integral_gpu_kernel <<<G1,B1>>> (d_activity, currentCamPointer, background_activity);

	CUDA_SAFE_CALL(hipDeviceSynchronize());
}


