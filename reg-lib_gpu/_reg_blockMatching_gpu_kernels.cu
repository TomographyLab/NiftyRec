#include "hip/hip_runtime.h"
/*
 *  _reg_blockMatching_kernels.cu
 *  
 *
 *  Created by Marc Modat and Pankaj Daga on 24/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef __REG_BLOCKMATCHING_KERNELS_CU__
#define __REG_BLOCKMATCHING_KERNELS_CU__

// Some parameters that we need for the kernel execution.
// The caller is supposed to ensure that the values are set

// Number of blocks in each dimension
__device__ __constant__ int3 c_BlockDim;
__device__ __constant__ int c_StepSize;
__device__ __constant__ int3 c_ImageSize;
__device__ __constant__ float r1c1;

// Transformation matrix from nifti header
__device__ __constant__ float4 t_m_a;
__device__ __constant__ float4 t_m_b;
__device__ __constant__ float4 t_m_c;

#define BLOCK_WIDTH 4
#define BLOCK_SIZE 64
#define OVERLAP_SIZE 3
#define STEP_SIZE 1

#include "_reg_blockMatching_gpu.h"

texture<float, 1, hipReadModeElementType> targetImageArray_texture;
texture<float, 1, hipReadModeElementType> resultImageArray_texture;
texture<int, 1, hipReadModeElementType> activeBlock_texture;

// Apply the transformation matrix
__device__ inline void apply_affine(const float4 &pt, float * result)
{    
	float4 mat = t_m_a;    
	result[0] = (mat.x * pt.x) + (mat.y*pt.y) + (mat.z*pt.z) + (mat.w);
	mat = t_m_b;
	result[1] = (mat.x * pt.x) + (mat.y*pt.y) + (mat.z*pt.z) + (mat.w);
	mat = t_m_c;
	result[2] = (mat.x * pt.x) + (mat.y*pt.y) + (mat.z*pt.z) + (mat.w);
}

// CUDA kernel to process the target values
__global__ void process_target_blocks_gpu(float *targetPosition_d,                                          
                                          float *targetValues)                                          
{   
    const int tid = (blockIdx.x * blockDim.x + threadIdx.x) + (blockIdx.y * gridDim.x);
    const int3 bDim = c_BlockDim;
	if (tid < bDim.x * bDim.y * bDim.z){
        const int currentBlockIndex = tex1Dfetch(activeBlock_texture,tid);        
        if (currentBlockIndex >= 0){
	        // Get the corresponding (i, j, k) indices	        
            int tempIndex = currentBlockIndex;
	        const int k =(int)(tempIndex/(bDim.x * bDim.y));
	        tempIndex -= k * bDim.x * bDim.y;
	        const int j =(int)(tempIndex/(bDim.x));
	        const int i = tempIndex - j * (bDim.x);
            const int offset = tid * BLOCK_SIZE;
		    const int targetIndex_start_x = i * BLOCK_WIDTH;
		    const int targetIndex_start_y = j * BLOCK_WIDTH;
		    const int targetIndex_start_z = k * BLOCK_WIDTH;
    		
		    int targetIndex_end_x = targetIndex_start_x + BLOCK_WIDTH;
		    int targetIndex_end_y = targetIndex_start_y + BLOCK_WIDTH;
		    int targetIndex_end_z = targetIndex_start_z + BLOCK_WIDTH;

            const int3 imageSize = c_ImageSize;
            const int threadId = tid % NUM_BLOCKS_TO_COMPARE;
            if (targetIndex_end_z > imageSize.z){                
                targetIndex_end_z = imageSize.z;
            }

            __shared__ int rampY[Block_target_block];
            rampY[threadId] = 0;
            if (targetIndex_end_y > imageSize.y){
                rampY[threadId] = targetIndex_end_y - imageSize.y;
                targetIndex_end_y = imageSize.y;
            }
            
            __shared__ int rampX[Block_target_block];
            rampX[threadId] = 0;
            if (targetIndex_end_x > imageSize.x){
                rampX[threadId] = targetIndex_end_x - imageSize.x;
                targetIndex_end_x = imageSize.x;
            }

            for (int count = 0; count < BLOCK_SIZE; ++count)
            {
                targetValues[count + offset] = 0.0f;
            }

		    unsigned int index = 0;	            
		    for(int z = targetIndex_start_z; z< targetIndex_end_z; ++z){                
		        int indexZ = z * imageSize.x * imageSize.y;
		        for(int y = targetIndex_start_y; y < targetIndex_end_y; ++y){                    
			        int indexXYZ = indexZ + y * imageSize.x + targetIndex_start_x;
                    for(int x = targetIndex_start_x; x < targetIndex_end_x; ++x){
				        targetValues[index + offset] = tex1Dfetch(targetImageArray_texture, indexXYZ);
			            indexXYZ++;
			            index++;
			        }
                    index += rampX[threadId];
		        }
                index += rampY[threadId] * BLOCK_WIDTH;
		    }

            float4 targetPosition;
		    targetPosition.x = i * BLOCK_WIDTH;
		    targetPosition.y = j * BLOCK_WIDTH;
		    targetPosition.z = k * BLOCK_WIDTH;        		    
            apply_affine(targetPosition, &(targetPosition_d[tid * 3]));            
        }
    }
}


// CUDA kernel to process the result blocks
__global__ void process_result_blocks_gpu(float *resultPosition_d,
                                          float *targetValues)
{    
    const int tid = (blockIdx.x * blockDim.x + threadIdx.x) + (blockIdx.y * gridDim.x);    
    const int3 bDim = c_BlockDim;
    const int ctid = (int)(tid / NUM_BLOCKS_TO_COMPARE);
    __shared__ float4 localCC [NUM_BLOCKS_TO_COMPARE];
    localCC[tid % NUM_BLOCKS_TO_COMPARE] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    __shared__ int updateThreadID;
    updateThreadID = -1;
    if (ctid < bDim.x * bDim.y * bDim.z) {
        const int activeBlockIndex = tex1Dfetch(activeBlock_texture, ctid);        
        int tempIndex = activeBlockIndex;
        const int k =(int)(tempIndex/(bDim.x * bDim.y));
	    tempIndex -= k * bDim.x * bDim.y;
	    const int j =(int)(tempIndex/(bDim.x));
	    const int i = tempIndex - j * (bDim.x);		        
		const int targetIndex_start_x = i * BLOCK_WIDTH;
		const int targetIndex_start_y = j * BLOCK_WIDTH;
		const int targetIndex_start_z = k * BLOCK_WIDTH;			
        
        if (activeBlockIndex >= 0) {            
            const int block_offset = ctid * BLOCK_SIZE;
            const int3 imageSize = c_ImageSize;
            tempIndex = tid % NUM_BLOCKS_TO_COMPARE;
            int n = (int)tempIndex /NUM_BLOCKS_TO_COMPARE_2D;
            tempIndex -= n * NUM_BLOCKS_TO_COMPARE_2D;            
            int m = (int)tempIndex /NUM_BLOCKS_TO_COMPARE_1D;
            int l = tempIndex - m * NUM_BLOCKS_TO_COMPARE_1D;
            n -= OVERLAP_SIZE;
            m -= OVERLAP_SIZE;
            l -= OVERLAP_SIZE;

            tempIndex = tid % NUM_BLOCKS_TO_COMPARE;

            int resultIndex_start_z = targetIndex_start_z + n;
            int resultIndex_end_z = resultIndex_start_z + BLOCK_WIDTH;

            int rampZ = 0;            
            if (resultIndex_start_z < 0){
                rampZ = -resultIndex_start_z;
                resultIndex_start_z = 0;                    
            }
            if (resultIndex_end_z > imageSize.z){
                resultIndex_end_z = imageSize.z;
            }

            int resultIndex_start_y = targetIndex_start_y + m;
			int resultIndex_end_y = resultIndex_start_y + BLOCK_WIDTH;
            int rampYLeft = 0;
            int rampYRight = 0;
            if (resultIndex_start_y < 0) {
                rampYLeft = -resultIndex_start_y;
                resultIndex_start_y = 0;
            }

            if (resultIndex_end_y > imageSize.y) {
                rampYRight = resultIndex_end_y - imageSize.y;
                resultIndex_end_y = imageSize.y;
            }

            int resultIndex_start_x = targetIndex_start_x + l;
			int resultIndex_end_x = resultIndex_start_x + BLOCK_WIDTH;
            int rampXLeft = 0;
            int rampXRight = 0;
            if (resultIndex_start_x < 0) {
                rampXLeft = -resultIndex_start_x;
                resultIndex_start_x = 0;
            }
            if (resultIndex_end_x > imageSize.x) {
                rampXRight = resultIndex_end_x - imageSize.x;
                resultIndex_end_x = imageSize.x;
            }            

            float target_mean = 0.0f;            
            float result_mean = 0.0f;            
            float voxel_number = 0.0f;
            float result_var = 0.0f;
            float target_var = 0.0f;
            float target_temp = 0.0f;
            float result_temp = 0.0f;
            float current_value = 0.0f;
            float current_target_value = 0.0f;

            localCC[tempIndex].w = 0.0f;
            __shared__ unsigned int index[NUM_BLOCKS_TO_COMPARE];
            index[tempIndex]= rampZ * BLOCK_WIDTH * BLOCK_WIDTH;
            for(int z = resultIndex_start_z; z< resultIndex_end_z; ++z){
                int indexZ = z * imageSize.y * imageSize.x;
                index[tempIndex] += rampYLeft * BLOCK_WIDTH;
                for(int y = resultIndex_start_y; y < resultIndex_end_y; ++y){                                    
                    int indexXYZ = indexZ + y * imageSize.x + resultIndex_start_x;
                    index[tempIndex] += rampXLeft;                                
                    for(int x = resultIndex_start_x; x < resultIndex_end_x; ++x){
                        current_value = tex1Dfetch(resultImageArray_texture, indexXYZ);
                        current_target_value = targetValues[block_offset + index[tempIndex]];
                        if (current_value != 0.0f && current_target_value != 0.0f) {
                            result_mean += current_value;
                            target_mean += current_target_value;
                            ++voxel_number;
                        }
                        indexXYZ++;
                        index[tempIndex]++;
                    }
                    index[tempIndex] += rampXRight;                                    
                }
                index[tempIndex] += rampYRight * BLOCK_WIDTH;                            
            }

            if (voxel_number > 0.0f) {
                result_mean /= voxel_number;
                target_mean /= voxel_number;
            }

            index[tempIndex] = rampZ * BLOCK_WIDTH * BLOCK_WIDTH;
            for(int z = resultIndex_start_z; z< resultIndex_end_z; ++z){
                int indexZ = z * imageSize.y * imageSize.x;
                index[tempIndex] += rampYLeft * BLOCK_WIDTH;
                for(int y = resultIndex_start_y; y < resultIndex_end_y; ++y){                                    
                    int indexXYZ = indexZ + y * imageSize.x + resultIndex_start_x;
                    index[tempIndex] += rampXLeft;                                
                    for(int x = resultIndex_start_x; x < resultIndex_end_x; ++x){
                        current_value = tex1Dfetch(resultImageArray_texture, indexXYZ);
                        current_target_value = targetValues[block_offset + index[tempIndex]];
                        if (current_value != 0.0f && current_target_value != 0.0f) {
                            target_temp = (current_target_value - target_mean);
                            result_temp = (current_value - result_mean);
                            result_var += result_temp * result_temp;
                            target_var += target_temp * target_temp;
                            localCC[tempIndex].w += target_temp * result_temp;
                        }
                        indexXYZ++;
                        index[tempIndex]++;
                    }
                    index[tempIndex] += rampXRight;                                    
                }
                index[tempIndex] += rampYRight * BLOCK_WIDTH;                            
            }            
            localCC[tempIndex].x = l;
            localCC[tempIndex].y = m;
            localCC[tempIndex].z = n;
            
            if (voxel_number > 0.0f) {
                target_var = sqrt(target_var/voxel_number);
                result_var = sqrt(result_var/voxel_number);
                if (target_var > 0.0f && result_var > 0.0f)
			        localCC[tempIndex].w = fabsf(localCC[tempIndex].w/
				        (voxel_number*target_var*result_var));
            }
            // Just take ownership of updating the final value
            if (updateThreadID == -1)
                updateThreadID = tid;
        }

        __syncthreads();
        // Just let one thread do the final update
        if (updateThreadID > -1) {
            float4 bestCC = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
            __shared__ unsigned dex[NUM_BLOCKS_TO_COMPARE];
            for (dex[tempIndex] = 0; dex[tempIndex] < NUM_BLOCKS_TO_COMPARE; ++dex[tempIndex]) {
                if (localCC[dex[tempIndex]].w > bestCC.w) {
                    bestCC.x = localCC[dex[tempIndex]].x;
                    bestCC.y = localCC[dex[tempIndex]].y;
                    bestCC.z = localCC[dex[tempIndex]].z;
                    bestCC.w = localCC[dex[tempIndex]].w;
                }
            }
            bestCC.x += targetIndex_start_x;
            bestCC.y += targetIndex_start_y;
            bestCC.z += targetIndex_start_z;            
            apply_affine(bestCC, &(resultPosition_d[ctid * 3]));
        }
    }
}

#endif

