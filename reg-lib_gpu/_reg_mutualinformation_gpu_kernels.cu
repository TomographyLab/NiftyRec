#include "hip/hip_runtime.h"
/*
 *  _reg_mutualinformation_kernels.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_MUTUALINFORMATION_kernels_CU
#define _REG_MUTUALINFORMATION_kernels_CU

__device__ __constant__ int c_VoxelNumber;
__device__ __constant__ int3 c_ImageSize;
__device__ __constant__ int c_Binning;
__device__ __constant__ float4 c_Entropies;
__device__ __constant__ float c_NMI;
__device__ __constant__ int c_ActiveVoxelNumber;

texture<float, 1, hipReadModeElementType> targetImageTexture;
texture<float, 1, hipReadModeElementType> resultImageTexture;
texture<float4, 1, hipReadModeElementType> resultImageGradientTexture;
texture<float, 1, hipReadModeElementType> histogramTexture;
texture<float4, 1, hipReadModeElementType> gradientImageTexture;
texture<int, 1, hipReadModeElementType> maskTexture;

__device__ float GetBasisSplineValue(float x)
{
	x=fabsf(x);
	float value=0.0f;
	if(x<2.0f)
		if(x<1.0f)
			value = 2.0f/3.0f + (0.5f*x-1.0f)*x*x;
		else{
			x-=2.0f;
			value = -x*x*x/6.0f;
	}
	return value;
}
__device__ float GetBasisSplineDerivativeValue(float ori)
{
	float x=fabsf(ori);
	float value=0.0f;
	if(x<2.0f)
		if(x<1.0f)
			value = (1.5f*x-2.0f)*ori;
		else{
			x-=2.0f;
			value = -0.5f * x * x;
			if(ori<0.0f)value =-value;
	}
	return value;
}

__global__ void reg_getVoxelBasedNMIGradientUsingPW_kernel(float4 *voxelNMIGradientArray_d)
{
	const int tid=blockIdx.x*blockDim.x+threadIdx.x;
	if(tid<c_ActiveVoxelNumber){

        const int targetIndex = tex1Dfetch(maskTexture,tid);
		float targetImageValue = tex1Dfetch(targetImageTexture,targetIndex);
		float resultImageValue = tex1Dfetch(resultImageTexture,targetIndex);
		float4 resultImageGradient = tex1Dfetch(resultImageGradientTexture,tid);
		
		float4 gradValue = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

		// No computation is performed if any of the point is part of the background
        // The two is added because the image is resample between 2 and bin +2
        // if 64 bins are used the histogram will have 68 bins et the image will be between 2 and 65
		if( targetImageValue>2.0f &&
            resultImageValue>2.0f){

            targetImageValue = floor(targetImageValue); // Parzen window filling of the joint histogram is approximated
            resultImageValue = floor(resultImageValue);

			float3 resDeriv = make_float3(
				resultImageGradient.x,
				resultImageGradient.y,
				resultImageGradient.z);
					
			float jointEntropyDerivative_X = 0.0f;
			float movingEntropyDerivative_X = 0.0f;
			float fixedEntropyDerivative_X = 0.0f;
					
			float jointEntropyDerivative_Y = 0.0f;
			float movingEntropyDerivative_Y = 0.0f;
			float fixedEntropyDerivative_Y = 0.0f;
					
			float jointEntropyDerivative_Z = 0.0f;
			float movingEntropyDerivative_Z = 0.0f;
			float fixedEntropyDerivative_Z = 0.0f;
					
			for(int t=(int)(targetImageValue-1.0f); t<(int)(targetImageValue+2.0f); t++){
				if(-1<t && t<c_Binning){
					for(int r=(int)(resultImageValue-1.0f); r<(int)(resultImageValue+2.0f); r++){
						if(-1<r && r<c_Binning){
							float commonValue = GetBasisSplineValue((float)t-targetImageValue) *
								GetBasisSplineDerivativeValue((float)r-resultImageValue);

							float jointLog = tex1Dfetch(histogramTexture, t*c_Binning+r);
							float targetLog = tex1Dfetch(histogramTexture, c_Binning*c_Binning+t);
							float resultLog = tex1Dfetch(histogramTexture, c_Binning*c_Binning+c_Binning+r);

							float temp = commonValue * resDeriv.x;
							jointEntropyDerivative_X -= temp * jointLog;
							fixedEntropyDerivative_X -= temp * targetLog;
							movingEntropyDerivative_X -= temp * resultLog;

							temp = commonValue * resDeriv.y;
							jointEntropyDerivative_Y -= temp * jointLog;
							fixedEntropyDerivative_Y -= temp * targetLog;
							movingEntropyDerivative_Y -= temp * resultLog;

							temp = commonValue * resDeriv.z;
							jointEntropyDerivative_Z -= temp * jointLog;
							fixedEntropyDerivative_Z -= temp * targetLog;
							movingEntropyDerivative_Z -= temp * resultLog;
						} // O<t<bin
					} // t
				} // 0<r<bin
			} // r

			float NMI= c_NMI;
            float temp = c_Entropies.z;
            // (Marc) I removed the normalisation by the voxel number as each gradient has to be normalised in the same way
			gradValue.x = (fixedEntropyDerivative_X + movingEntropyDerivative_X - NMI * jointEntropyDerivative_X) / temp;
			gradValue.y = (fixedEntropyDerivative_Y + movingEntropyDerivative_Y - NMI * jointEntropyDerivative_Y) / temp;
			gradValue.z = (fixedEntropyDerivative_Z + movingEntropyDerivative_Z - NMI * jointEntropyDerivative_Z) / temp;

		}
		voxelNMIGradientArray_d[targetIndex]=gradValue;

	}
	return;
}

#endif
